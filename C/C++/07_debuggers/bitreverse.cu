
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 8

__global__ void bitreverse(unsigned int *data) {
    int idx = threadIdx.x;
    unsigned int val = data[idx];
    data[idx] = ((val & 0xf0) >> 4) | ((val & 0x0f) << 4);
}

int main() {
    unsigned int h_data[N];
    for (int i = 0; i < N; i++) h_data[i] = i;
    unsigned int *d_data;
    hipMalloc(&d_data, N * sizeof(unsigned int));
    hipMemcpy(d_data, h_data, N * sizeof(unsigned int), hipMemcpyHostToDevice);

    bitreverse<<<1, N>>>(d_data);
    hipDeviceSynchronize();

    hipMemcpy(h_data, d_data, N * sizeof(unsigned int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) printf("%u -> %u\n", i, h_data[i]);

    hipFree(d_data);
    return 0;
}
